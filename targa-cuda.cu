// vanessa writes a targa file
// compiling and running this program will produce a targa file
//
// compile : $ gcc create-tga-from-any-input.c -o targa-exe
// usage   : $ ./targa-exe input-file output-filename dimensions
// example : $ ./targa-exe /usr/input/filename.txt /usr/output/filename.tga 1000
//           ==> filename.tga that is 1000 x 1000 pixels large
//
// a lot of this is based on Grant Emery's file https://www.tjhsst.edu/~dhyatt/superap/code/targa.c thanks dude
// author: vanessa pyne --- github.com/vipyne




#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <string.h>
#include <errno.h>
#include <math.h>

#define BYTE_RANGE 256
#define RGBA 3 // 3 for RGB, 4 for RGBA
#define PI 3.14159265358979323846

////// targa file header

typedef struct {
char id_length;       // length of id field (number of bytes - max 255)
char map_type;        // colormap field (0 or 1; no map or 256 entry palette)
char image_type;      // ( 0 - no image data included
                      //   1 - uncompressed, color mapped image
                      //   2 - uncompressed, RGB image
                      //   3 - uncompressed, black & white image
                      //   9 - run-length encoded(RLE-lossless compression),color mapped image
                      //  10 - RLE, RGB image
                      //  11 - compressed, black & white image )

  int map_first;       // first entry index for color map
  int map_length;      // total number of entries in color map
  char map_entry_size; // number of bits per entry

  int x;               // x cooridinate of origin
  int y;               // y cooridinate of origin

  int width;           // width in pixels
  int height;          // height in pixels

  char bits_per_pixel; // number of bits per pixel

  char misc;           // srsly? "scan origin and alpha bits" this example uses scan origin
                       // honestly, don't know what's going on here. we pass in a hex value
                       // :shrug_emoji:
} targa_header;

int little_endianify (int number)
{
  return number % BYTE_RANGE;
}

int big_endianify (int number)
{
  return number / BYTE_RANGE;
}

// used if sorting the pixels
int compare_function (const void* a_pointer, const void* b_pointer)
{
  return *(( char* )a_pointer) - *(( char* )b_pointer);
}

////// write header function

void write_header (targa_header header, FILE *tga)
{
  fputc( header.id_length, tga );
  fputc( header.map_type, tga );
  fputc( header.image_type, tga );

  fputc( little_endianify(header.map_first), tga );
  fputc( big_endianify(header.map_first), tga );

  fputc( little_endianify(header.map_length), tga );
  fputc( big_endianify(header.map_length), tga );

  fputc( header.map_entry_size, tga );

  fputc( little_endianify(header.x), tga );
  fputc( big_endianify(header.x), tga );
  fputc( little_endianify(header.y), tga );
  fputc( big_endianify(header.y), tga );

  fputc( little_endianify(header.width), tga );
  fputc( big_endianify(header.width), tga );
  fputc( little_endianify(header.height), tga );
  fputc( big_endianify(header.height), tga );

  fputc( header.bits_per_pixel, tga );
  fputc( header.misc, tga );
}

void print_directions(void)
{
  printf("$ ./targa-exe input-file output-filename dimension\n");
}


////// CUDA KERNEL
__global__
void thisIsBasicallyAShaderInMyBook(int n, char *gpu_normalized_input, char *gpu_normalized_sorted, char *gpu_output)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // if (i < n) y[i] = a*x[i] + y[i];
  if (i < n) {
    if (i % 2 == 0) {
      gpu_output[i] = 0x21;
    } else {
		  gpu_output[i] = gpu_normalized_input[i];
    }
    //printf("gpu-output: %c\n",gpu_output[i]);
  }

	// //printf("^^^^ writing pixels \n");
 //  int n_index = 0;
	// float theta = 0.0;

 //  //// magic happens here
 //  for (int y = 0; y < HEIGHT; ++y)
 //  {
 //    for (int x = 0; x < WIDTH; ++x)
 //    {
 //      // pixels read in B G R order
 //      fputc( fabsf((x%200)-(y)/(float)sin(x-y) ), tga); ////// BLUE

	// 		float butter = sin(theta*10.0)* (250.0-y) - (float)sin(theta/250.0)*200.0 - 200.0;

	// 		if (y > butter+(x-50)*(float)log(100) ) { ////// IF
	// 			n_index++;
	// 			fputc(normalized_input[n_index] + (float)sin(n_index*theta)-y/5, tga); ///// GREEN
	// 			fputc(normalized_sorted[n_index]+(float)log(normalized_sorted[n_index]*500)-x/40, tga); ///// RED
	// 		} else {
	// 			n_index--;
	// 			fputc((normalized_sorted[n_index]) + ((x-y)/(float)brain), tga); ////// GREEN
	// 			n_index--;
	// 			fputc((normalized_input[n_index] - (float)sin(theta/10.0) - x - y/2) / 3.7 + butter/3, tga); ///// RED
	// 		}
 //      n_index++;
	// 		theta+=0.001;
 //    }
 //  }
 //  //// magic ends here
}


////// MAIN

int main (int argc, char* argv[])
{
  if (argc != 4) {
    printf("\n");
    printf("Please enter correct number of arguments. --\n");
    print_directions();
    printf("\n");
    return 1;
  }

  FILE *source;
  source = fopen(argv[1], "rb");

  if (source == NULL) {
    printf("Source file `%s` cannot be found. --\n", argv[1]);
    return 1;
  }

  FILE *tga;                    // pointer to file that we will write
  tga = fopen(argv[2], "wbx");  // `x` needed for `errno` to work
  int overwrite_warning = errno;

  if (overwrite_warning != 0) {
    printf("Destination file `%s` already exists. --\n", argv[2]);
    return 1;
  }

  int HEIGHT = atoi(argv[3]);
  int WIDTH = atoi(argv[3]);

  if (errno != 0) {
    fclose(tga);
    unlink(argv[2]);

    printf("`%s` is not a valid dimension. Please use a number. --\n", argv[3]);
    return 1;
  }

  // intialize and set TARGA header values
  targa_header header;       // variable of targa_header type

  // int x, y;                  // coordinates for `for` loops to pass in
                             // correct number of pixel values

  header.id_length = 0;
  header.map_type = 0;
  header.image_type = 2;     // uncompressed RGB image

  header.map_first = 0;
  header.map_length = 0;
  header.map_entry_size = 0;

  header.x = 0;
  header.y = 0;
  header.width = WIDTH;
  header.height = HEIGHT;

  header.bits_per_pixel = 24;
  header.misc = 0x20;       // scan from upper left corner, need to investigate this further

  // start to write file
  write_header(header, tga);
  printf("^^^^ header written\n");

  // source input file

  fseek(source, 0, SEEK_END);
  int source_size = ftell(source);
  rewind(source);
  printf("^^^^ source file read,      length: %d\n", source_size);

  // width * height = number of pixels
  int input_binary_length = WIDTH * HEIGHT * RGBA; // normal people call this a buffer

  // buffer for pixel values (no zeros)
  char fart[input_binary_length];
  char normalized_input[input_binary_length];

	//char *normalized_input;
	//normalized_input = (char*)malloc(input_binary_length * sizeof(char));

	char *host_buffer;
  host_buffer = (char*)malloc(input_binary_length * sizeof(char));

  // buffer for sorted pixel values (still no zeros)
  char normalized_sorted[input_binary_length];

  // buffer for entire input file
  char *read_through = (char*) malloc ( sizeof(char) * source_size );


 // CUDA buffers
  int N = input_binary_length;
  char *gpu_output;
  char *gpu_normalized_input;
  char *gpu_normalized_sorted;

  hipMalloc(&gpu_output, N * sizeof(char));
  hipMalloc(&gpu_normalized_input, N * sizeof(char));
  //cudaMalloc(&gpu_normalized_sorted, N * sizeof(char));
  hipMalloc(&gpu_normalized_sorted,  N * sizeof(char));
  printf("^^^^ !! d      N                       : %d \n", N);
  printf("^^^^ !! d                  sizeof(char): %d \n", (int)sizeof(char));
  printf("^^^^ !! lu                 sizeof(char): %lu \n", sizeof(char));
  printf("^^^^ !! lu     sizeof(normalized_input): %lu \n", sizeof(normalized_input));
  printf("^^^^ !! lu sizeof(gpu_normalized_input): %lu \n", sizeof(*gpu_normalized_input));

  printf("N * sizeof(char) : %lu\n", N * sizeof(char) );
  //printf("input_binary_length / N : %d / %d\n", input_binary_length, N);







  int i = 0;
  int read_through_index = 0;
  fread(read_through, 1, source_size, source);
  printf("read_through : %s\n", read_through);
  printf("read_through : %lu\n", sizeof(read_through));
  printf("read_through : %lu\n", strlen(read_through));
  printf("read_through[0] : %c\n", read_through[0]);
  printf("read_through[421] : %c\n", read_through[421]);
  // strncpy(normalized_input, read_through, input_binary_length);
  while (i < input_binary_length)
  {
    normalized_input[i] = read_through[read_through_index];

    // fread(read_through, 1, source_size, source);
    // if (read_through_index >= source_size)
    // {
    //   //printf("rewinding\n");
    //   rewind(source);
    //   read_through_index = 0;
    // }
    // if (read_through[read_through_index] != '0')
    // {
    //   normalized_input[i] = read_through[read_through_index];
      i++;
    // }
    read_through_index++;
  }
  printf("^^^^ normalized buffer set, length: %d \n", (int) sizeof(normalized_input));

  // strncpy(normalized_sorted, normalized_input, input_binary_length);
  // qsort(normalized_sorted, strlen(normalized_input), sizeof(char), compare_function);


  for (int i = 0; i < input_binary_length; ++i) {
    printf("bah %d\n", i);
    // normalized_input[i] = i;
    fart[i] = i;
    printf("bah %d\n", fart[i]);

  }


  hipMemcpy(gpu_output, host_buffer, N * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(gpu_normalized_sorted, &normalized_sorted, N * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(gpu_normalized_input, &normalized_input, N * sizeof(char), hipMemcpyHostToDevice);
  //cudaMemcpy(gpu_normalized_sorted, &normalized_sorted, N * sizeof(char), cudaMemcpyHostToDevice);

  //printf("^^^^ CUDA input buffer set, length: %lu \n", sizeof(gpu_normalized_input));
  //printf("^^^^ CUDA sorted buffer set, length: %lu \n", sizeof(gpu_normalized_sorted));
  //printf("^^^^ CUDA output buffer set, length: %lu \n", sizeof(gpu_output));
  free(read_through);




	// Magic here / kernel / just a shader ////////////////////////
	// kernal_name <<< `execution configuration` >>> (args)
	// <<< grid dimensions (optional), block dimensions / # of thread blocks in grid, # of threads in thread block >>>
	thisIsBasicallyAShaderInMyBook <<< (N+255)/256, 256 >>>(N, gpu_normalized_input, gpu_normalized_sorted, gpu_output);
	////////////// <<< >>> //////////////////




	hipMemcpy(host_buffer, gpu_output, N * sizeof(char), hipMemcpyDeviceToHost);

	printf("size of host buffer : %lu\n", sizeof(host_buffer));






  fputs(host_buffer, tga); //////////////////////////


   // int n_index = 0;
   // for (int y = 0; y < HEIGHT; ++y)
   // {
   //   for (int x = 0; x < WIDTH; ++x)
   //  {
			// fputc(normalized_input[n_index],tga);
			// fputc(normalized_input[n_index],tga);
			// fputc(normalized_input[n_index],tga);
   //     n_index++;
   //   }
   // }
 //  //// magic ends here  //////////////////////////






	//for (int i = 0; i < 1000; ++i){
		//fputc(33, tga);
		//printf("host_buffer: %c", host_buffer[i]);
	//}
	printf("\n");





  hipFree(gpu_output);
  hipFree(gpu_normalized_input);
  hipFree(gpu_normalized_sorted);

  free(host_buffer);
	//free(normalized_input);

	fclose(tga);
  fclose(source);
  printf("^^^^ finished! marvel at your targa!\n");

  return 0;
}
